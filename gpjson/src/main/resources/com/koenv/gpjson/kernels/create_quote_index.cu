
#include <hip/hip_runtime.h>
__global__ void create_quote_index(char *file, long n, long *escape_index, long *quote_index, char *quote_carry_index, long quote_index_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  // We want to always calculate on 64-character boundaries, such that we can put
  // all bits of 64 characters into 1 long.
  long normal_chars_per_thread = (n+stride-1) / stride;
  long chars_per_thread = ((normal_chars_per_thread + 64 - 1) / 64) * 64;
  long start = index * chars_per_thread;
  long end = start + chars_per_thread;

  // This will contain the bitmask of escaped characters
  long escaped = 0;

  // Temporary variable for storing the current bit index
  long bit_index = 0;

  int quote_count = 0;

  int final_loop_iteration = end;
  if (n < end) {
    final_loop_iteration = n;
  }

  for (long i = start; i < final_loop_iteration; i += 1) {
    long offsetInBlock = i % 64;

    // At the start of each boundary (including the first), set the escaped characters
    if (offsetInBlock == 0) {
      escaped = escape_index[i / 64];
    }

    if (file[i] == '"') {
      if ((escaped & (1L << offsetInBlock)) == 0) {
        bit_index = bit_index | (1L << offsetInBlock);
        quote_count++;
      }
    }

    // If we are at the end of a boundary, set our result. We do not do it
    // if we are at the end since that would reset our bit_index.
    if (offsetInBlock == 63L) {
      quote_index[i / 64] = bit_index;
      // Reset the bit index since we're starting over
      bit_index = 0;
    }
  }

  if (n < end && (final_loop_iteration - 1) % 64 != 63L && n - start > 0) {
    // In the final thread with data, we need to do this to make sure the last longs are actually set
    int final_index = (final_loop_iteration - 1) / 64;
    quote_index[final_index] = bit_index;
  }

  quote_carry_index[index] = quote_count & 1;
}
