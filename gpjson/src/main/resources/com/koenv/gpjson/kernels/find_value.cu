
#include <hip/hip_runtime.h>
__global__ void find_value(char *file, long n, long *new_line_index, long new_line_index_size, long *string_index, long *leveled_bitmaps_index, long leveled_bitmaps_index_size, long level_size, int num_levels, long *result) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  long lines_per_thread = (new_line_index_size+stride-1) / stride;

  long start = index * lines_per_thread;
  long end = start + lines_per_thread;

  int current_level = 0;
  char *looking_for = "user";
  int looking_for_length = 4;

  for (long i = start; i < end && i < new_line_index_size; i += 1) {
    result[i] = -1;
    long new_line_start = new_line_index[i];
    long new_line_end = (i + 1 < new_line_index_size) ? new_line_index[i+1] : n;

    for (long j = new_line_start; j < new_line_end && j < n; j += 1) {
      bool is_structural = (leveled_bitmaps_index[level_size * current_level + j / 64] & (1L << j % 64)) != 0;

      if (is_structural && file[j] == ':') {
        // Start looking for the end of the string
        long string_end_index = -1;
        for (long k = j - 1; k > new_line_start; k -= 1) {
          if ((string_index[k / 64] & (1L << k % 64)) != 0) {
            string_end_index = k;
            break;
          }
        }

        assert(string_end_index >= new_line_start);

        long string_start_index = string_end_index - looking_for_length;
        if (string_start_index < new_line_start || file[string_start_index] != '"') {
          continue;
        }

        bool found = true;

        for (long k = 0; k < looking_for_length; k++) {
          if (looking_for[k] != file[string_start_index + k + 1]) {
            found = false;
            break;
          }
        }

        if (!found) {
          continue;
        }

        // This means we are at the correct key, so we'll increase our level
        if (current_level == 0) {
          current_level++;
          looking_for = "lang";
        } else if (current_level == 1) {
          result[i] = j;
          break;
        }
      }
    }
  }
}
